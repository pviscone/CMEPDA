#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void kernel(void){
    printf("Hello, World from the GPU! (block: %d thread:(%d %d %d))\n",blockIdx.x,threadIdx.x,threadIdx.y,threadIdx.z);
}

int main(){
    kernel<<<2,2>>>();
    hipDeviceSynchronize();
    cout << "Hello, World from the CPU!" << endl;
    return 0;
}