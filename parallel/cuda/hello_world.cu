#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

//definizione del kernel
__global__ void kernel(void){
    printf("Hello, World from the GPU! (block: %d thread:(%d %d %d))\n",blockIdx.x,threadIdx.x,threadIdx.y,threadIdx.z);
}

int main(){
    //kernel lanciato su 2 blocchi di 2 thread (eseguito 4 volte)
    kernel<<<2,2>>>();
    hipDeviceSynchronize();
    cout << "Hello, World from the CPU!" << endl;
    return 0;
}