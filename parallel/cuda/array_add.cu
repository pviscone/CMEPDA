#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include <stdio.h>
#include <vector>
#include <iostream>

using namespace std;

/* Creo degli array nella global memory della GPU, li sommo
e alla fine li copio in un array nella host memory

Potrei usare i metodi di thrust per evitare i loop per
scrivere e leggere sul vettore*/

__global__ void ker (float * a, float *b, float *c, size_t n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        a[idx]=idx;
        b[idx]=2*idx;
        c[idx] = a[idx] + b[idx];
    }
}


int main(){
    //In questo caso converrebbe istanziare i device_vector direttamente sulla GPU senza fare la copia dall'host ma
    // è giusto per fare un esempio
    vector <float> h_a{1,2,3,4,5,6,7,8,9,10};
    vector <float> h_b{2,3,4,5,6,7,8,9,10,11};
    size_t n = h_a.size();
    thrust::device_vector<float> a(n), b(n),c(n);
    thrust::host_vector<float> host_res(n) ;

    //Copia gli std::vector nei vettori della GPU
    a=h_a;
    b=h_b;

    //Alloca il numero minimo di threads necessari per soddisfare il task
    int num_blocks = 1+n/1024;
    int num_threads = (n/num_blocks)+1;


    ker<<<num_blocks,num_threads>>>(a.data().get(),b.data().get(),c.data().get(),n);
    hipDeviceSynchronize();

    //copia il risultato che è nella GPU nell'host
    host_res = c;
    for (int i = 0; i < n; i++)
    {
        cout << host_res[i] << endl;
    }
    return 0;
}
