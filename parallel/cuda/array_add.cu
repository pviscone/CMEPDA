#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include <stdio.h>
#include <vector>
#include <iostream>
#include <numeric>

using namespace std;

/* Creo degli array nella global memory della GPU, li sommo
e alla fine li copio in un array nella host memory

Potrei usare i metodi di thrust per evitare i loop per
scrivere e leggere sul vettore*/

__global__ void ker (float * a, float *b, float *c, size_t n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        a[idx]=idx;
        b[idx]=2*idx;
        c[idx] = a[idx] + b[idx];
    }
}

int main(){
    //In questo caso converrebbe istanziare i device_vector direttamente sulla GPU senza fare la copia dall'host ma
    // è giusto per fare un esempio
    vector <float> h_a(1e4);
    vector <float> h_b(1e4);
    //creo vettori da 0 a 1e4-1 e da 2 a 1e4+1
    iota(h_a.begin(), h_a.end(), 0);
    iota(h_b.begin(), h_b.end(), 2);
    size_t n = h_a.size();
    thrust::device_vector<float> a(n), b(n),c(n);
    thrust::host_vector<float> host_res(n) ;

    //Copia gli std::vector nei vettori della GPU
    a=h_a;
    b=h_b;

    //Alloca il numero minimo di threads necessari per soddisfare il task
    const int thread_per_block=1024;
    const int num_blocks = n/thread_per_block+1;


    ker<<<num_blocks,thread_per_block>>>(a.data().get(),b.data().get(),c.data().get(),n);
    hipDeviceSynchronize();

    //copia il risultato che è nella GPU nell'host
    host_res = c;
    for (int i = 0; i < n; i++)
    {
        cout << host_res[i] << endl;
    }
    return 0;
}
